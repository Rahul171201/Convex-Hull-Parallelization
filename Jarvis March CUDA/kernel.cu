#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <chrono>
#include <fstream>

using namespace std;
using namespace std::chrono;

#define N 1024      // number of points in the plane
#define MAX_VAL 1000 // maximum value of any point in the plane

int bottom_most_point_x = 0;
int bottom_most_point_y = 0;

// Device function To find orientation of points p,q and r
// The function returns following values
// 0 --> p, q and r are collinear
// 1 --> Clockwise
// 2 --> Counterclockwise
__device__ int orientation(int p_x, int p_y, int q_x, int q_y, int r_x, int r_y)
{
    int val = (q_y - p_y) * (r_x - q_x) - (q_x - p_x) * (r_y - q_y);

    if (val == 0)
        return 0;
    return (val > 0) ? 1 : 2;
}

// Function to get the bottom most point
int getStartingPoint(int* x, int* y, int n)
{
    int min_y = INT_MAX, min_x = INT_MAX;
    int index = -1;
    for (int i = 0; i < n; i++)
    {
        if (y[i] < min_y)
        {
            min_y = y[i];
            min_x = x[i];
            index = i;
        }
        else if (y[i] == min_y)
        {
            if (min_x > x[i])
                index = i;
            min_x = min(min_x, x[i]);
        }
    }
    return index;
}

// Kernel function to find the minimum counter clockwise angle
__global__ void min_angle(int* x, int* y, int p, int n, int* temp_point)
{
    int i = threadIdx.x; // get thread id
    int stride = 1; // stride
    int temp_rem = 2;

    if (i >= n)
        return;

    __syncthreads();

    while (stride < n) {
        if (i % temp_rem == 0) {
            if (orientation(x[p], y[p], x[temp_point[i + stride]], y[temp_point[i + stride]], x[temp_point[i]], y[temp_point[i]]) == 2) {
                temp_point[i] = temp_point[i + stride];
            }
        }

        temp_rem = temp_rem * 2;
        stride = stride * 2;

        __syncthreads(); // synchronization barrier
    }
}

int sequential_orientation(int p_x, int p_y, int q_x, int q_y, int r_x, int r_y)
{
    int val = (q_y - p_y) * (r_x - q_x) - (q_x - p_x) * (r_y - q_y);

    if (val == 0)
        return 0;
    return (val > 0) ? 1 : 2;
}

// Function that defines the Jarvis March algorithm
void sequentialJarvisMarch(int* x, int* y, int n)
{
    // There must be at least 3 points for a convex hull to be possible
    if (n < 3)
    {
        cout << "Convex Hull cannot be formed\n";
        exit(0);
    }

    int* hull_x = (int*)malloc(n * sizeof(int)); // stores the convex hull points
    int* hull_y = (int*)malloc(n * sizeof(int));

    // Find the bottommost point
    int starting_point = getStartingPoint(x, y, n);
    int count = 0;
    int p = starting_point, q;
    do
    {
        // Add current point to the convex hull
        hull_x[count] = x[p];
        hull_y[count] = y[p];
        count++;
        q = (p + 1) % n; // Let's say q is the most counter clockwise point

        for (int i = 0; i < n; i++)
        {
            // If i is more counterclockwise than current q, then update q
            if (sequential_orientation(x[p], y[p], x[i], y[i], x[q], y[q]) == 2)
                q = i;
        }

        p = q;

    } while (p != starting_point); // Repeat the process until we again reach the starting point

    // cout<<"The convex hull points after sequential algoroithm are:\n";
    // for(int i=0;i<n;i++){
    //     cout<<hull_x[i]<<" "<<hull_y[i]<<"\n";
    // }
}

// Main function
int main()
{
    srand(time(0));

    int* x;
    int* y;

    // unified shared memory
    hipMallocManaged(&x, N * sizeof(int));
    hipMallocManaged(&y, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        int val_x = (rand() % (MAX_VAL - 1 + 1)) + 1;
        int val_y = (rand() % (MAX_VAL - 1 + 1)) + 1;
        x[i] = val_x;
        y[i] = val_y;
    }

    // Sequential Convex Hull Computation
    auto start = high_resolution_clock::now();
    sequentialJarvisMarch(x, y, N);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);

    cout << "Time taken by sequential algorithm = " << duration.count() << " microseconds" << endl;

    // Parallel Convex Hull Computation

    // There must be at least 3 points for a convex hull to be possible
    if (N < 3)
    {
        cout << "Convex hull is not possible, since we require more than 3 points\n";
        exit(0);
    }

    // Initialize the hull
    int* hull_x = (int*)malloc(N * sizeof(int));
    int* hull_y = (int*)malloc(N * sizeof(int));

    int* temp_point;

    hipMallocManaged(&temp_point, N * sizeof(int));

    // Find the bottommost starting point
    int starting_point = getStartingPoint(x, y, N);

    start = high_resolution_clock::now();
    int p = starting_point;
    int count = 0;
    do
    {
        // Add current point to convex hull
        hull_x[count] = x[p];
        hull_y[count] = y[p];
        count++;

        for (int i = 0; i < N; i++) {
            temp_point[i] = i;
        }

        min_angle << <1, N >> > (x, y, p, N, temp_point);
        hipDeviceSynchronize();

        p = temp_point[0];

    } while (p != starting_point); // Repeat the process until and unless we reach ths starting point
    stop = high_resolution_clock::now();
    duration = duration_cast<microseconds>(stop - start);

    cout << "Time taken by parallel algorithm = " << duration.count() << " microseconds" << endl;

    int* convex_hull = (int*)malloc(2 * N * sizeof(int));

    for (int i = 0; i < N; i++) {
        convex_hull[2 * i] = hull_x[i];
        convex_hull[(2 * i + 1)] = hull_y[i];
    }

    // cout << "The points are:\n";
    // for (int i = 0; i < N; i++)
    // {
    //     cout << "{" << x[i] << ", " << y[i] << "}\n";
    // }

    // cout << "The convex hull points are:\n";
    // for (int i = 0; i < 2 * count; i = i + 2) {
    //     cout << convex_hull[i] << " " << convex_hull[i + 1] << "\n";
    // }

    // write the output onto a file
    freopen("points.txt", "w", stdout);

    cout << N << "\n";

    for (int i = 0; i < N; i++)
    {
        cout << x[i] << " " << y[i] << "\n";
    }

    cout << count << "\n";

    for (int i = 0; i < 2 * count; i = i + 2)
    {
        cout << convex_hull[i] << " " << convex_hull[i + 1] << "\n";
    }

    // free memory
    free(hull_x);
    free(hull_y);
    hipFree(temp_point);
    hipFree(temp_point);
    hipFree(temp_point);
}